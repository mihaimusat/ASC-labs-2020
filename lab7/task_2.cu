#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "utils/utils.h"

// TODO 6: Write the code to add the two arrays element by element and 
// store the result in another array
__global__ void add_arrays(const float *a, const float *b, float *c, int N) {
	
	unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i < N) {
		c[i] = a[i] + b[i];
	}   
}

int main(void) {
    	hipSetDevice(0);
    	int N = 1 << 20;
	const int num_bytes = N * sizeof(float);
	hipError_t error;

	const size_t block_size = 256;
	size_t num_blocks;
	
    	float *host_array_a = 0;
   	float *host_array_b = 0;
    	float *host_array_c = 0;

    	float *device_array_a = 0;
    	float *device_array_b = 0;
    	float *device_array_c = 0;

    	// TODO 1: Allocate the host's arrays
	// TODO 2: Allocate the device's arrays
        // TODO 3: Check for allocation errors

	host_array_a = (float *) malloc(num_bytes);
        DIE(host_array_a == NULL, "malloc");

	host_array_b = (float *) malloc(num_bytes);
        DIE(host_array_b == NULL, "malloc");

	host_array_c = (float *) malloc(num_bytes);
        DIE(host_array_c == NULL, "malloc");

        error = hipMalloc(&device_array_a, num_bytes);
        DIE(error != hipSuccess || device_array_a == NULL, "hipMalloc");

	error = hipMalloc(&device_array_b, num_bytes);
        DIE(error != hipSuccess || device_array_b == NULL, "hipMalloc");

	error = hipMalloc(&device_array_c, num_bytes);
        DIE(error != hipSuccess || device_array_c == NULL, "hipMalloc");	
	

    	// TODO 4: Fill array with values; use fill_array_float to fill
    	// host_array_a and fill_array_random to fill host_array_b. Each
    	// function has the signature (float *a, int n), where n = number of elements.
	
	fill_array_float(host_array_a, N);
	fill_array_random(host_array_b, N);
	
    	// TODO 5: Copy the host's arrays to device
	error = hipMemcpy(device_array_a, host_array_a,
		num_bytes, hipMemcpyHostToDevice);
	DIE(error != hipSuccess, "hipMemcpy(host_array_a)");

	error = hipMemcpy(device_array_b, host_array_b,
		num_bytes, hipMemcpyHostToDevice);
	DIE(error != hipSuccess, "hipMemcpy(host_array_b)");

    	// TODO 6: Execute the kernel, calculating first the grid size
    	// and the amount of threads in each block from the grid
    	// Hint: For this execise the block_size can have any value lower than the
    	//      API's maximum value (it's recommended to be close to the maximum
    	//      value).

	num_blocks = N / block_size;

	if (N % block_size) {
		++num_blocks;
	}

	add_arrays<<<num_blocks, block_size>>>(device_array_a, device_array_b,
		device_array_c, N);

	error = hipDeviceSynchronize();
	DIE(error != hipSuccess, "hipDeviceSynchronize");

    	// TODO 7: Copy back the results and then uncomment the checking function

	error = hipMemcpy(host_array_c, device_array_c,
		num_bytes, hipMemcpyDeviceToHost);
	DIE(error != hipSuccess, "hipMemcpy(device_array_c)");
    	
	check_task_2(host_array_a, host_array_b, host_array_c, N);

    	// TODO 8: Free the memory
	free(host_array_a);
	free(host_array_b);
	free(host_array_c);

	error = hipFree(device_array_a);
	DIE(error != hipSuccess, "hipFree(device_array_a)");

	error = hipFree(device_array_b);
	DIE(error != hipSuccess, "hipFree(device_array_b)");

	error = hipFree(device_array_c);
	DIE(error != hipSuccess, "hipFree(device_array_c)");
   
    	return 0;
}
